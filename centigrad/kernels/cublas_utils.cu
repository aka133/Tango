
#include <hip/hip_runtime.h>
#include <hipblas.h>

/*
    Compute: C = α(A @ B) + βC using cuBLAS SGEMM
    
    Args:
        A: Input matrix (M, K) - must be contiguous GPU tensor
        B: Input matrix (K, N) - must be contiguous GPU tensor
        C: Output matrix (M, N) - must be contiguous GPU tensor
        M, N, K: Matrix dimensions
        alpha: Scalar for AB product (default: 1.0)
        beta: Scalar for C (default: 0.0)
        handle: Existing cuBLAS handle (optional)
*/

extern "C" {
    void cublas_matmul(float* A, float* B, float* C,
                        int M, int N, int K,
                        float alpha, float beta) {
        
        hipblasHandle_t handle; 
        hipblasCreate(&handle);

        // Tell CUBLAS we're using row-major ordering
        hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH);
        hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);
        
        // For row-major: C = A @ B
        // In CUBLAS (column-major): C = B^T @ A^T
        hipblasSgemm(handle, 
                    HIPBLAS_OP_N, HIPBLAS_OP_N,    // No transpositions needed
                    N, M, K,                      // Dimensions for column-major result
                    &alpha, 
                    B, K,                         // B is KxN
                    A, K,                         // A is MxK
                    &beta, 
                    C, N);                        // C is MxN

        hipblasDestroy(handle);
    }
}